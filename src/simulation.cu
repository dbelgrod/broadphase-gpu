#include "hip/hip_runtime.h"
#include <gpubf/simulation.cuh>
#include <gpubf/queue.cuh>


#include <thrust/sort.h>
#include <thrust/execution_policy.h>

using namespace ccdgpu;

#include <cmath>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void setup(int devId, int& smemSize, int& threads, int& nboxes);



void run_collision_counter(Aabb* boxes, int N) {

    // int N = 200000;
    // Aabb boxes[N];
    // for (int i = 0; i<N; i++)
    // {
    //     boxes[i] = Aabb(i);
    //     // printf("box %i created\n", boxes[i].id);
    // }

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_counter;
    hipMalloc((void**)&d_counter, sizeof(int));
    reset_counter<<<1,1>>>(d_counter);
    hipDeviceSynchronize();

     int collisions;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    // int bytes_mem_intrfce = 352 >> 3;
    // int mem_clock_rate = 1376 << 1;
    // float bandwidth_mem_theor = (mem_clock_rate * bytes_mem_intrfce) / pow(10, 3);

    // Set up timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Get number of collisions
    hipEventRecord(start);
    count_collisions<<<1,1>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("(count_collisions<<<1,1>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    printf("Effective Bandwidth (GB/s): %.6f (GB/s)\n", 32*2/milliseconds/1e6);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<1,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<1,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<2,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<2,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.6f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<56,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<56,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);

    reset_counter<<<1,1>>>(d_counter);
    hipEventRecord(start);
    count_collisions<<<256,1024>>>(d_boxes, d_counter, N); 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(&collisions, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n(count_collisions<<<256,1024>>>)\n");
    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Elapsed time: %.9f ms/c\n", milliseconds/collisions);
    printf("Collision: %i\n", collisions);
    return;
    // printf("%zu\n", sizeof(Aabb));


    // Retrieve count from GPU and print out
    // int counter;
    // hipMemcpy(&counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    // printf("count: %d\n", counter);
    // return 0;
}

void run_scaling(const Aabb* boxes,  int N, int desiredBoxesPerThread, vector<unsigned long>& finOverlaps)
{

    int devId = 0;
    hipSetDevice(devId);

    int smemSize;
    int threads;

    setup(devId, smemSize, threads, desiredBoxesPerThread);
    const int nBoxesPerThread = desiredBoxesPerThread ? desiredBoxesPerThread : smemSize / sizeof(Aabb) / (2*(BLOCK_PADDED));
    printf("Boxes per Thread: %i\n", nBoxesPerThread);

    finOverlaps.clear();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    // guess overlaps size
    int guess = 0;

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // Allocate counter to GPU + set to 0 collisions
    int * d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    reset_counter<<<1,1>>>(d_count);
    hipDeviceSynchronize();

    //Count collisions
    count_collisions<<<1,1>>>(d_boxes, d_count, N); 
    int count;
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    reset_counter<<<1,1>>>(d_count);
    printf("Total collisions from counting: %i\n", count);



    int2 * d_overlaps;
    hipMalloc((void**)&d_overlaps, sizeof(int2)*(guess));

    dim3 block(BLOCK_SIZE_1D,BLOCK_SIZE_1D);
    // dim3 grid ( (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D,  (N+BLOCK_SIZE_1D)/BLOCK_SIZE_1D );
    int grid_dim_1d = (N+BLOCK_SIZE_1D)/ BLOCK_SIZE_1D / nBoxesPerThread;
    dim3 grid( grid_dim_1d, grid_dim_1d );
    printf("Grid dim (1D): %i\n", grid_dim_1d);
    printf("Box size: %i\n", sizeof(Aabb));

    long long * d_queries;
    hipMalloc((void**)&d_queries, sizeof(long long)*(1));
    reset_counter<<<1,1>>>(d_queries);

    printf("Shared mem alloc: %i B\n", nBoxesPerThread*2*(BLOCK_PADDED)*sizeof(Aabb) );
    hipEventRecord(start);
    get_collision_pairs<<<grid, block, nBoxesPerThread*2*(BLOCK_PADDED)*sizeof(Aabb)>>>(d_boxes, d_count, d_overlaps, N, guess, nBoxesPerThread, d_queries);
    // get_collision_pairs_old<<<grid, block>>>(d_boxes, d_count, d_overlaps, N, guess);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // hipDeviceSynchronize();

    long long queries;
    hipMemcpy(&queries, d_queries, sizeof(long long), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("queries: %llu\n", queries);
    printf("needed queries: %llu\n", (long long)N*(N-1)/2 );

    // int count;
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    

    if (count > guess) //we went over
    {
        printf("Running again\n");
        hipFree(d_overlaps);
        hipMalloc((void**)&d_overlaps, sizeof(int2)*(count));
        reset_counter<<<1,1>>>(d_count);
        hipDeviceSynchronize();
        hipEventRecord(start);
        get_collision_pairs<<<grid, block, nBoxesPerThread*2*(BLOCK_PADDED)*sizeof(Aabb)>>>(d_boxes, d_count, d_overlaps, N, count, nBoxesPerThread, d_queries);
        // get_collision_pairs_old<<<grid, block>>>(d_boxes, d_count, d_overlaps, N, 2*count);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        // hipDeviceSynchronize();
    }

    printf("Elapsed time: %.6f ms\n", milliseconds);
    printf("Collisions: %i\n", count);
    printf("Elapsed time: %.9f ms/collision\n", milliseconds/count);
    printf("Boxes: %i\n", N);
    printf("Elapsed time: %.9f ms/box\n", milliseconds/N);
    // printf("Elapsed time: %.15f us/query\n", (milliseconds*1000)/((long long)N*N/2));

    int2 * overlaps =  (int2*)malloc(sizeof(int2) * (count));
    gpuErrchk(hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost));


    hipFree(d_overlaps);
    // for (size_t i=0; i< count; i++)
    // {
    //     // finOverlaps.push_back(overlaps[i].x, overlaps[i].y);
    //     // finOverlaps.push_back(overlaps[i].y);
        
    //     const Aabb& a = boxes[overlaps[i].x];
    //     const Aabb& b = boxes[overlaps[i].y];
    //     if (a.type == Simplex::VERTEX && b.type == Simplex::FACE)
    //     {
    //         finOverlaps.push_back(a.ref_id);
    //         finOverlaps.push_back(b.ref_id);
    //     }
    //     else if (a.type == Simplex::FACE && b.type == Simplex::VERTEX)
    //     {
    //         finOverlaps.push_back(b.ref_id);
    //         finOverlaps.push_back(a.ref_id);
    //     }
    //     else if (a.type == Simplex::EDGE && b.type == Simplex::EDGE)
    //     {   
    //         finOverlaps.push_back(min(a.ref_id, b.ref_id));
    //         finOverlaps.push_back(max(a.ref_id, b.ref_id));
    //     }
    // }

    printf("Total(filt.) overlaps: %lu\n", finOverlaps.size() / 2);
    free(overlaps);
    // free(counter);
    // free(counter);
    hipFree(d_count);
    hipDeviceReset();

}
//  // // //////// / / // / / // / // // / //  /

struct sort_sweepmarker_x
{
    __host__ __device__
    bool operator()(const SweepMarker &a, const SweepMarker &b) const {
    return (a.x < b.x);}
};


struct sorter{};

struct sort_aabb_x : sorter 
{
    __host__ __device__
    bool operator()(const Aabb &a, const Aabb &b) const {
    return (a.min.x < b.min.x);}

    __host__ __device__
    bool operator()(const float3 &a, const float3 &b) const {
        return (a.x < b.x);}

    // __host__ __device__
    // bool operator()(const SortedMin &a, const SortedMin &b) const {
    //     return (a.min < b.min);}
    
    __host__ __device__
    bool operator()(const RankBox &a, const RankBox &b) const {
        return (a.aabb->min.x < b.aabb->min.x);}

};

// struct sort_aabb_y : sorter 
// {
//      // useless bc x is alays the min
//     // __host__ __device__
//     // bool operator()(const float3 &a, const float3 &b) const {
//     //     return (a.y < b.y);}
    
//     __host__ __device__
//     bool operator()(const RankBox &a, const RankBox &b) const {
//         return (a.aabb->min.y < b.aabb->min.y);}

// };

// struct sort_aabb_z : sorter
// {
//     // useless bc x is alays the min
//     // __host__ __device__
//     // bool operator()(const float3 &a, const float3 &b) const {
//     //     return (a.z < b.z);}

//     __host__ __device__
//     bool operator()(const RankBox &a, const RankBox &b) const {
//         return (a.aabb->min.z < b.aabb->min.z);}
    
// };

// struct sort_cantor
// {
//     __host__ __device__
//     bool operator()(const RankBox &a, const RankBox &b) const {
//         return (a.rank_c < b.rank_c);}
// };


// DEPRECATED
// void run_sweep(const Aabb* boxes, int N, int nbox, vector<pair<int,int>>& finOverlaps, int& threads)
// {
//     int devId = 0;
//     hipSetDevice(devId);

//     int smemSize;

//     setup(devId, smemSize, threads, nbox);

//     finOverlaps.clear();
//     hipEvent_t start, stop;
//     hipEventCreate(&start);
//     hipEventCreate(&stop); 

//     // Allocate boxes to GPU 
//     Aabb * d_boxes;
//     hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
//     hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

//     // Allocate counter to GPU + set to 0 collisions
//     int * d_count;
//     hipMalloc((void**)&d_count, sizeof(int));
//     reset_counter<<<1,1>>>(d_count);
//     hipDeviceSynchronize();

//     dim3 block(threads);
//     int grid_dim_1d = (N / threads + 1); 
//     dim3 grid( grid_dim_1d );
//     printf("Grid dim (1D): %i\n", grid_dim_1d);
//     printf("Box size: %i\n", sizeof(Aabb));
//     printf("SweepMarker size: %i\n", sizeof(SweepMarker));

//     // int* d_index;
//     // hipMalloc((void**)&d_index, sizeof(int)*(N));
//     int* rank;
//     hipMalloc((void**)&rank, sizeof(int)*(1*N));

//     int* rank_x = &rank[0];
//     // int* rank_y = &rank[N];
//     // int* rank_z = &rank[2*N];

//     // Translate boxes -> SweepMarkers
//     hipEventRecord(start);
//     build_index<<<grid,block>>>(d_boxes, N, rank_x);
//     // build_index<<<grid,block>>>(d_boxes, N, rank_y);
//     // build_index<<<grid,block>>>(d_boxes, N, rank_z);
//     hipEventRecord(stop);
//     hipEventSynchronize(stop);
//     float milliseconds = 0;
//     hipEventElapsedTime(&milliseconds, start, stop);

//     printf("Elapsed time for build: %.6f ms\n", milliseconds);

//     // Thrust sort (can be improved by sort_by_key)
//     hipEventRecord(start);
//     // thrust::sort(thrust::device, d_axis, d_axis + N, sort_sweepmarker_x() );
//     try{
//         thrust::sort_by_key(thrust::device, d_boxes, d_boxes + N, rank_x, sort_aabb_x() );
//         }
//     catch (thrust::system_error &e){
//         printf("Error: %s \n",e.what());}
    
//     hipEventRecord(stop);
//     hipEventSynchronize(stop);
//     milliseconds = 0;
//     hipEventElapsedTime(&milliseconds, start, stop);

//     printf("Elapsed time for sort: %.6f ms\n", milliseconds);

//     // Test print some sorted output
//     // print_sort_axis<<<1,1>>>(d_boxes,rank_x, 5);
//     hipDeviceSynchronize();

//     // Find overlapping pairs
//     int guess = 0;
//     int2 * d_overlaps;
//     hipMalloc((void**)&d_overlaps, sizeof(int2)*(guess));

//     int count;
//     retrieve_collision_pairs<<<grid, block, smemSize>>>(d_boxes, rank_x, d_count, d_overlaps, N, guess, nbox);
//     hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
//     hipDeviceSynchronize();

//     if (count > guess) //we went over
//     {
//         printf("Running again\n");
//         hipFree(d_overlaps);
//         hipMalloc((void**)&d_overlaps, sizeof(int2)*(count));
//         reset_counter<<<1,1>>>(d_count);
//         hipDeviceSynchronize();
//         hipEventRecord(start);
//         retrieve_collision_pairs<<<grid, block, smemSize>>>(d_boxes, rank_x, d_count, d_overlaps, N, count, nbox);
//         hipEventRecord(stop);
//         hipEventSynchronize(stop);
//         print_overlap_start<<<1,1>>>(d_overlaps); 
//         hipDeviceSynchronize();
//         milliseconds = 0;
//         hipEventElapsedTime(&milliseconds, start, stop);
//         printf("Elapsed time for findoverlaps: %.6f ms\n", milliseconds);
//     }
//     // int count;
//     hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
//     hipDeviceSynchronize();

//     printf("Elapsed time: %.6f ms\n", milliseconds);
//     printf("Collisions: %i\n", count);
//     printf("Elapsed time: %.9f ms/collision\n", milliseconds/count);
//     printf("Boxes: %i\n", N);
//     printf("Elapsed time: %.9f ms/box\n", milliseconds/N);

//     int2 * overlaps =  (int2*)malloc(sizeof(int2) * (count));
//     hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost);

//     printf("Final count: %i\n", count);

//     hipFree(d_overlaps);
//     for (size_t i=0; i < count; i++)
//     {
//         finOverlaps.emplace_back(overlaps[i].x, overlaps[i].y);
//         // finOverlaps.push_back(overlaps[i].y);
        
//         // const Aabb& a = boxes[overlaps[i].x];
//         // const Aabb& b = boxes[overlaps[i].y];
//         // if (a.type == Simplex::VERTEX && b.type == Simplex::FACE)
//         // {
//         //     finOverlaps.emplace_back(a.ref_id, b.ref_id);
//         // }
//         // else if (a.type == Simplex::FACE && b.type == Simplex::VERTEX)
//         // {
//         //     finOverlaps.emplace_back(b.ref_id, a.ref_id);
//         // }
//         // else if (a.type == Simplex::EDGE && b.type == Simplex::EDGE)
//         // {
//         //     finOverlaps.emplace_back(min(a.ref_id, b.ref_id), max(a.ref_id, b.ref_id));
//         // }
//     }

//     printf("Total(filt.) overlaps: %lu\n", finOverlaps.size() );
//     free(overlaps);
//     // free(counter);
//     // free(counter);
//     hipFree(d_overlaps);
//     hipFree(d_count); 

//     hipDeviceReset();
// }

// MULTI GPU SWEEP SUPPORT
void merge_local_overlaps(
    const tbb::enumerable_thread_specific<tbb::concurrent_vector<std::pair<int,int>>>& storages,
    std::vector<std::pair<int,int>>& overlaps)
{
    overlaps.clear();
    size_t num_overlaps = overlaps.size();
    for (const auto& local_overlaps : storages) {
        num_overlaps += local_overlaps.size();
    }
    // serial merge!
    overlaps.reserve(num_overlaps);
    for (const auto& local_overlaps : storages) {
        overlaps.insert(
            overlaps.end(), local_overlaps.begin(), local_overlaps.end());
    }
}

void run_sweep_multigpu(const Aabb* boxes, int N, int nbox, vector<pair<int, int>>& finOverlaps, int& threads, int & devcount)
{
    cout<<"default threads "<<tbb::task_scheduler_init::default_num_threads()<<endl;
    tbb::enumerable_thread_specific<tbb::concurrent_vector<pair<int,int>>> storages;

    float milliseconds = 0;
    int device_init_id = 0;

    int smemSize;
    setup(device_init_id, smemSize, threads, nbox);

    hipSetDevice(device_init_id);

    finOverlaps.clear();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    dim3 block(threads);
    int grid_dim_1d = (N / threads + 1); 
    dim3 grid( grid_dim_1d );
    printf("Grid dim (1D): %i\n", grid_dim_1d);
    printf("Box size: %i\n", sizeof(Aabb));

    // int* rank;
    // hipMalloc((void**)&rank, sizeof(int)*(1*N));

    // int* rank_x = &rank[0];
    // int* rank_y = &rank[N];
    // int* rank_z = &rank[2*N];

    // Translate boxes -> SweepMarkers

    // hipEventRecord(start);
    // build_index<<<grid,block>>>(d_boxes, N, rank_x);
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    
    // hipEventElapsedTime(&milliseconds, start, stop);

    // printf("Elapsed time for build: %.6f ms\n", milliseconds);

    // Thrust sort (can be improved by sort_by_key)
    hipEventRecord(start);
    try{
        // thrust::sort_by_key(thrust::device, d_boxes, d_boxes + N, rank_x, sort_aabb_x() );
        thrust::sort(thrust::device, d_boxes, d_boxes + N, sort_aabb_x());
        }
    catch (thrust::system_error &e){
        printf("Error: %s \n",e.what());}
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time for sort: %.6f ms\n", milliseconds);


    // Test print some sorted output
    // print_sort_axis<<<1,1>>>(d_boxes, 5);
    hipDeviceSynchronize();
    

    int devices_count;
    hipGetDeviceCount(&devices_count);
    // devices_count-=2;
    devices_count = devcount ? devcount : devices_count;
    int range = ceil( (float)N / devices_count); 

    // free(start);
    // free(stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipEvent_t starts[devices_count];
    hipEvent_t stops[devices_count];
    float millisecondss[devices_count];

    tbb::parallel_for(0, devices_count, 1, [&](int & device_id)    {

        
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device_id);
        printf("%s -> unifiedAddressing = %d\n", prop.name, prop.unifiedAddressing);

        hipSetDevice(device_id);

        // hipEvent_t start, stop;
        hipEventCreate(&starts[device_id]);
        hipEventCreate(&stops[device_id]); 

        int is_able;

        hipDeviceCanAccessPeer(&is_able, device_id, device_init_id);
        hipDeviceSynchronize();
        if (is_able)
        { 
            hipDeviceEnablePeerAccess(device_init_id, 0); 
            hipDeviceSynchronize(); 
        }
        else if (device_init_id != device_id)
            printf("Device %i cant access Device %i\n", device_id, device_init_id);

        int range_start  = range*device_id;
        int range_end = range*(device_id + 1);
        printf("device_id: %i [%i, %i)\n", device_id, range_start, range_end);
        

        Aabb * d_b;
        hipMalloc((void**)&d_b, sizeof(Aabb)*N);
        hipMemcpy(d_b, d_boxes, sizeof(Aabb)*N, hipMemcpyDefault);
        hipDeviceSynchronize();  
       
        hipDeviceCanAccessPeer(&is_able, device_id, device_init_id);
        hipDeviceSynchronize();
        if (is_able)
        { 
            hipDeviceDisablePeerAccess(device_init_id); 
            hipDeviceSynchronize();  
        }
        else if (device_init_id != device_id)
            printf("Device %i cant access Device %i\n", device_id, device_init_id);

        
        // Allocate counter to GPU + set to 0 collisions
        int * d_count;
        gpuErrchk(hipMalloc((void**)&d_count, sizeof(int)));
        gpuErrchk(hipMemset(d_count, 0, sizeof(int)));
        gpuErrchk( hipGetLastError() );   

        // Find overlapping pairs
        int guess = N*360;
        printf("Guess %i\n", guess);

        int2 * d_overlaps;
        hipMalloc((void**)&d_overlaps, sizeof(int2)*(guess));
        gpuErrchk( hipGetLastError() ); 
        
        int grid_dim_1d = (range / threads + 1); 
        dim3 grid( grid_dim_1d );

        int count;
        hipEventRecord(starts[device_id]);
        retrieve_collision_pairs<<<grid, block, smemSize>>>(d_b, d_count, d_overlaps, N, guess, nbox, range_start, range_end);
        hipEventRecord(stops[device_id]);
        hipEventSynchronize(stops[device_id]);
        hipEventElapsedTime(&millisecondss[device_id], starts[device_id], stops[device_id]);
        hipDeviceSynchronize();
        hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
        printf("count for device %i : %i\n", device_id, count);
    
        if (count > guess){
            printf("Running again\n");
            hipFree(d_overlaps);
            hipMalloc((void**)&d_overlaps, sizeof(int2)*(count));
            // hipMemset(d_overlaps, 0, sizeof(int2)*(count));
            hipMemset(d_count, 0, sizeof(int));
            hipEventRecord(starts[device_id]);
            retrieve_collision_pairs<<<grid, block, smemSize>>>(d_b, d_count, d_overlaps, N, count, nbox, range_start, range_end);
            hipEventRecord(stops[device_id]);
            hipEventSynchronize(stops[device_id]);
            hipEventElapsedTime(&millisecondss[device_id], starts[device_id], stops[device_id]);
            hipDeviceSynchronize();
            hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
            printf("count2 for device %i : %i\n", device_id, count);
        }

        
        // printf("Elapsed time: %.9f ms/collision\n", milliseconds/count);
        // printf("Boxes: %i\n", N);
        // printf("Elapsed time: %.9f ms/box\n", milliseconds/N);

        // int2 * overlaps = new int2[count];
        int2* overlaps =  (int2*)malloc(sizeof(int2) * count);
        gpuErrchk(hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost));
        gpuErrchk( hipGetLastError() ); 

        printf("Final count for device %i:  %i\n", device_id, count);

        auto& local_overlaps = storages.local();
        // local_overlaps.reserve(local_overlaps.size() + count);
        
        // auto is_face = [&](Aabb x){return x.vertexIds.z >= 0;};
        // auto is_edge = [&](Aabb x){return x.vertexIds.z < 0 && x.vertexIds.y >= 0 ;};
        // auto is_vertex = [&](Aabb x){return x.vertexIds.z < 0  && x.vertexIds.y < 0;};
        
        for (size_t i=0; i < count; i++)
        {
            // local_overlaps.emplace_back(overlaps[i].x, overlaps[i].y);
            // finOverlaps.push_back();
            int aid = overlaps[i].x;
            int bid = overlaps[i].y;
            Aabb a = boxes[aid];
            Aabb b = boxes[bid];
    
            if (is_vertex(a) && is_face(b)) //vertex, face
                local_overlaps.emplace_back(aid, bid);
            else if (is_edge(a) && is_edge(b))
                local_overlaps.emplace_back(min(aid, bid), max(aid,bid));
            else if (is_face(a) && is_vertex(b))
                local_overlaps.emplace_back(bid, aid);
        }
        
        
        printf("Total(filt.) overlaps for devid %i: %i\n", device_id, local_overlaps.size());
        // delete [] overlaps;
        // free(overlaps);
        
        // // free(counter);
        // // free(counter);
        // hipFree(d_overlaps);
        // hipFree(d_count); 
        // hipFree(d_b);
        // hipFree(d_r);
        // hipDeviceReset();

    }); //end tbb for loop

    merge_local_overlaps(storages, finOverlaps);

    float longest = 0;
    for (int i=0; i<devices_count; i++)
    {
        for (int j=0; j<devices_count; j++)
        {
            hipEventElapsedTime(&milliseconds, starts[i], stops[j]);
            longest = milliseconds > longest ? milliseconds : longest;
        }

    }
    printf("\n");
    printf("Elapsed time: %.6f ms\n", longest);
    printf("Merged overlaps: %i\n", finOverlaps.size());
    printf("\n");

}

void run_sweep_sharedqueue(const Aabb* boxes, int N, int nbox, vector<pair<int, int>>& finOverlaps, int2*& d_overlaps, int *& d_count, int& threads, int & devcount)
{
 
    int device_init_id = 0;

    int smemSize;
    setup(device_init_id, smemSize, threads, nbox);

    hipSetDevice(device_init_id);

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    dim3 block(threads);
    int grid_dim_1d = (N / threads + 1); 
    dim3 grid( grid_dim_1d );
    printf("Grid dim (1D): %i\n", grid_dim_1d);
    printf("Box size: %i\n", sizeof(Aabb));
    // printf("MiniBox size: %i\n", sizeof(MiniBox));
    printf("float3 size: %i\n", sizeof(float3));
    printf("sizeof(queue) size: %i\n", sizeof(Queue));
    
    
    // float3 * d_sortedmin;
    // hipMalloc((void**)&d_sortedmin, sizeof(float3)*N);
    float3 * d_sm;
    hipMalloc((void**)&d_sm, sizeof(float3)*N);

    MiniBox * d_mini;
    hipMalloc((void**)&d_mini, sizeof(MiniBox)*N);

    // mean of all box points (used to find best axis)
    float3 * d_mean;
    hipMalloc((void**)&d_mean, sizeof(float3));
    hipMemset(d_mean, 0, sizeof(float3));

    // recordLaunch("create_ds", grid_dim_1d, threads, smemSize, create_ds, d_boxes, d_sm, d_mini, N, d_mean);
    recordLaunch("calc_mean", grid_dim_1d, threads, smemSize, calc_mean, d_boxes, d_mean, N);

    // temporary
    float3 mean;
    hipMemcpy(&mean, d_mean, sizeof(float3), hipMemcpyDeviceToHost);
    printf("mean: x %.6f y %.6f z %.6f\n", mean.x, mean.y, mean.z);

    // calculate variance and determine which axis to sort on
    float3 * d_var; //2 vertices per box
    hipMalloc((void**)&d_var, sizeof(float3));
    hipMemset(d_var, 0, sizeof(float3));
    // calc_variance(boxes, d_var, N, d_mean);
    recordLaunch("calc_variance", grid_dim_1d, threads, smemSize, calc_variance, d_boxes, d_var, N, d_mean);
    hipDeviceSynchronize();

    float3 var3d;
    hipMemcpy(&var3d, d_var, sizeof(float3), hipMemcpyDeviceToHost);
    float maxVar = max(max(var3d.x, var3d.y), var3d.z);

    printf("var: x %.6f y %.6f z %.6f\n", var3d.x, var3d.y, var3d.z);

    Dimension axis;
    if (maxVar == var3d.x)
        axis = x;
    else if (maxVar == var3d.y)
        axis = y;
    else 
        axis = z;
    //hack
    axis = x;

    printf("Axis: %s\n", axis == x ? "x" : (axis == y ? "y" : "z"));

    recordLaunch("create_ds", grid_dim_1d, threads, smemSize, create_ds, d_boxes, d_sm, d_mini, N, axis);


    try{
        // thrust::sort(thrust::device, d_sortedmin, d_sortedmin + N, sort_aabb_x() );
        thrust::sort(thrust::device, d_sm, d_sm + N, sort_aabb_x());
        }
    catch (thrust::system_error &e){
        printf("Thrust error: %s \n",e.what());}
    
    gpuErrchk( hipGetLastError() );


    int count = 0*N;

    // int * d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    hipMemset(d_count, 0, sizeof(int));

    // int2 * d_overlaps;
    hipMalloc((void**)&d_overlaps, sizeof(int2)*count);

    recordLaunch<float3 *, const MiniBox *, int2 *, int, int *, int, int, int>("twostage_queue_1st", 2*grid_dim_1d, threads,twostage_queue, d_sm, d_mini, d_overlaps, N, d_count, count, 0, INT_MAX);
    gpuErrchk(hipDeviceSynchronize());
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("1st count for device %i:  %i\n", device_init_id, count);

    hipMalloc((void**)&d_overlaps, sizeof(int2)*(count)); 
    hipMemset(d_count, 0, sizeof(int));
    recordLaunch<float3 *, const MiniBox *, int2 *, int, int *, int, int, int>("twostage_queue_2nd", 2*grid_dim_1d, threads,twostage_queue, d_sm, d_mini, d_overlaps, N, d_count, count, 0, INT_MAX);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
    printf("Final count for device %i:  %i\n", device_init_id, count);
    // hipMalloc((void**)&d_overlaps, sizeof(int2)*(count)); //big enough
    // hipMemset(d_count, 0, sizeof(int));
    // retrieve_collision_pairs2<<<grid2, block, 49152>>>(d_boxes, d_count, outpair, d_overlaps, N, count);
    
    int2* overlaps =  (int2*)malloc(sizeof(int2) * count);
    gpuErrchk(hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost));
    gpuErrchk( hipGetLastError() ); 

    printf("Final count for device %i:  %i\n", 0, count);

    auto& local_overlaps = finOverlaps;
    // local_overlaps.reserve(local_overlaps.size() + count);
    
    hipFree(d_boxes);
    hipFree(d_mini);
    hipFree(d_sm);
    
    for (size_t i=0; i < count; i++)
    {
        // local_overlaps.emplace_back(overlaps[i].x, overlaps[i].y);
        // finOverlaps.push_back();
        int aid = overlaps[i].x;
        int bid = overlaps[i].y;
        Aabb a = boxes[aid];
        Aabb b = boxes[bid];

        if (is_vertex(a) && is_face(b)) //vertex, face
            local_overlaps.emplace_back(aid, bid);
        else if (is_edge(a) && is_edge(b))
            local_overlaps.emplace_back(min(aid, bid), max(aid,bid));
        else if (is_face(a) && is_vertex(b))
            local_overlaps.emplace_back(bid, aid);
        // if (is_vertex(a) && is_face(b)) //vertex, face
        // {
        //     local_overlaps.emplace_back(a.ref_id, b.ref_id);
        // }
        // else if (is_face(a) && is_vertex(b))
        // {
        //     local_overlaps.emplace_back(b.ref_id, a.ref_id);
        // }
        // else if (is_edge(a) && is_edge(b))
        // {
        //     local_overlaps.emplace_back(min(a.ref_id, b.ref_id), max(a.ref_id, b.ref_id));
        // }
    }
    free(overlaps);
    printf("Total(filt.) overlaps for devid %i: %i\n", 0, local_overlaps.size());
}


// void run_sweep_pairing(const Aabb* boxes, int N, int nbox, vector<pair<int, int>>& finOverlaps, int& threads, int & devcount)
// {
 
//     int device_init_id = 0;

//     int smemSize;
//     setup(device_init_id, smemSize, threads, nbox);

//     hipSetDevice(device_init_id);

//     // Allocate boxes to GPU 
//     Aabb * d_boxes;
//     hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
//     hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

//     int block= threads;
//     int grid = (N / threads + 1); 

//     RankBox * d_rankboxes;
//     hipMalloc((void**)&d_rankboxes, sizeof(RankBox)*N);

//     create_rankbox<<<grid, block>>>(d_boxes, d_rankboxes, N);
    
//     thrust::sort(thrust::device, d_rankboxes, d_rankboxes + N, sort_aabb_x() );
//     register_rank_x<<<grid, block>>>(d_rankboxes, N);

//     thrust::sort(thrust::device, d_rankboxes, d_rankboxes + N, sort_aabb_y() );
//     register_rank_y<<<grid, block>>>(d_rankboxes, N);

//     assign_rank_c<<<grid, block>>>(d_rankboxes, N);
//     thrust::sort(thrust::device, d_rankboxes, d_rankboxes + N, sort_cantor() );

//     // print_stats<<<1,1>>>(d_rankboxes, N);

//     int count = 0;

//     int * d_count;
//     hipMalloc((void**)&d_count, sizeof(int));
//     hipMemset(d_count, 0, sizeof(int));
    
//     int2 * d_overlaps;
//     hipMalloc((void**)&d_overlaps, sizeof(int2)*count);
//     printf("sizeof(RankBox): %i\n", sizeof(RankBox));
//     printf("sharedMem: %i\n", sizeof(RankBox)*block);
//     build_checker2<<<grid, block, 49152>>>(d_rankboxes, d_overlaps, N, d_count, count);
//     gpuErrchk(hipDeviceSynchronize());
//     gpuErrchk( hipGetLastError() );

//     hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
//     printf("First count from building all overlapping x: %i\n", count);
//     hipFree(d_overlaps);
//     gpuErrchk(hipMalloc((void**)&d_overlaps, sizeof(int2)*count));
//     gpuErrchk(hipMemset(d_count, 0, sizeof(int)));
//     gpuErrchk( hipGetLastError() ); 
//     recordLaunch<const RankBox *, int2 *, int, int *, int>("build_checker2", grid, block, 49152, build_checker2, d_rankboxes, d_overlaps, N, d_count, count);
//     // build_checker2<<<grid, block>>>(d_rankboxes, d_overlaps, N, d_count, count);

//     // build_checker<<<grid, block, 49152>>>(d_sortedmin, outpair, N, d_count, count);
//     gpuErrchk( hipDeviceSynchronize());
//     hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
//     printf("Second count from building all overlapping x: %i\n", count);
//     gpuErrchk( hipGetLastError() ); 
    
//     printf("Final count for device %i:  %i\n", device_init_id, count);

//      // int2 * overlaps = new int2[count];
//      int2* overlaps =  (int2*)malloc(sizeof(int2) * count);
//      gpuErrchk(hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost));
//      gpuErrchk( hipGetLastError() ); 

//      printf("Final count for device %i:  %i\n", 0, count);

//      auto& local_overlaps = finOverlaps;
//      // local_overlaps.reserve(local_overlaps.size() + count);
     
//      auto is_face = [&](Aabb x){return x.vertexIds.z >= 0;};
//      auto is_edge = [&](Aabb x){return x.vertexIds.z < 0 && x.vertexIds.y >= 0 ;};
//      auto is_vertex = [&](Aabb x){return x.vertexIds.z < 0  && x.vertexIds.y < 0;};
     
     
//     //  for (size_t i=0; i < count; i++)
//     //  {
//     //      // local_overlaps.emplace_back(overlaps[i].x, overlaps[i].y);
//     //      // finOverlaps.push_back();
         
//     //      Aabb a = boxes[overlaps[i].x];
//     //      Aabb b = boxes[overlaps[i].y];
         
//     //      if (is_vertex(a) && is_face(b)) //vertex, face
//     //      {
//     //          local_overlaps.emplace_back(a.ref_id, b.ref_id);
//     //      }
//     //      else if (is_face(a) && is_vertex(b))
//     //      {
//     //          local_overlaps.emplace_back(b.ref_id, a.ref_id);
//     //      }
//     //      else if (is_edge(a) && is_edge(b))
//     //      {
//     //          local_overlaps.emplace_back(min(a.ref_id, b.ref_id), max(a.ref_id, b.ref_id));
//     //      }
//     //  }
     
//      printf("Total(filt.) overlaps for devid %i: %i\n", 0, local_overlaps.size());
// }

void run_sweep_multigpu_queue(const Aabb* boxes, int N, int nbox, vector<pair<int, int>>& finOverlaps, int& threads, int & devcount)
{
    cout<<"default threads "<<tbb::task_scheduler_init::default_num_threads()<<endl;
    tbb::enumerable_thread_specific<tbb::concurrent_vector<pair<int,int>>> storages;

    float milliseconds = 0;
    int device_init_id = 0;

    int smemSize;
    setup(device_init_id, smemSize, threads, nbox);

    hipSetDevice(device_init_id);

    finOverlaps.clear();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    dim3 block(threads);
    int grid_dim_1d = (N / threads + 1); 
    dim3 grid( grid_dim_1d );
    printf("Grid dim (1D): %i\n", grid_dim_1d);
    printf("Box size: %i\n", sizeof(Aabb));

    float3 * d_sm;
    hipMalloc((void**)&d_sm, sizeof(float3)*N);

    MiniBox * d_mini;
    hipMalloc((void**)&d_mini, sizeof(MiniBox)*N);

    // mean of all box points (used to find best axis)
    float3 * d_mean;
    hipMalloc((void**)&d_mean, sizeof(float3));
    hipMemset(d_mean, 0, sizeof(float3));

     // recordLaunch("create_ds", grid_dim_1d, threads, smemSize, create_ds, d_boxes, d_sm, d_mini, N, d_mean);
     recordLaunch("calc_mean", grid_dim_1d, threads, smemSize, calc_mean, d_boxes, d_mean, N);

     // temporary
     float3 mean;
     hipMemcpy(&mean, d_mean, sizeof(float3), hipMemcpyDeviceToHost);
     printf("mean: x %.6f y %.6f z %.6f\n", mean.x, mean.y, mean.z);
 
     // calculate variance and determine which axis to sort on
     float3 * d_var; //2 vertices per box
     hipMalloc((void**)&d_var, sizeof(float3));
     hipMemset(d_var, 0, sizeof(float3));
     // calc_variance(boxes, d_var, N, d_mean);
     recordLaunch("calc_variance", grid_dim_1d, threads, smemSize, calc_variance, d_boxes, d_var, N, d_mean);
     hipDeviceSynchronize();
 
     float3 var3d;
     hipMemcpy(&var3d, d_var, sizeof(float3), hipMemcpyDeviceToHost);
     float maxVar = max(max(var3d.x, var3d.y), var3d.z);
 
     printf("var: x %.6f y %.6f z %.6f\n", var3d.x, var3d.y, var3d.z);
 
     Dimension axis;
     if (maxVar == var3d.x)
         axis = x;
     else if (maxVar == var3d.y)
         axis = y;
     else 
         axis = z;
 
     printf("Axis: %s\n", axis == x ? "x" : (axis == y ? "y" : "z"));
 
     recordLaunch("create_ds", grid_dim_1d, threads, smemSize, create_ds, d_boxes, d_sm, d_mini, N, axis);
 
 
     try{
         // thrust::sort(thrust::device, d_sortedmin, d_sortedmin + N, sort_aabb_x() );
         thrust::sort(thrust::device, d_sm, d_sm + N, sort_aabb_x());
         }
     catch (thrust::system_error &e){
         printf("Thrust error: %s \n",e.what());}
     
    gpuErrchk( hipGetLastError() );
    hipDeviceSynchronize();
    

    int devices_count;
    hipGetDeviceCount(&devices_count);
    // devices_count-=2;
    devices_count = devcount ? devcount : devices_count;
    int range = ceil( (float)N / devices_count); 

    // free(start);
    // free(stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipEvent_t starts[devices_count];
    hipEvent_t stops[devices_count];
    float millisecondss[devices_count];

    tbb::parallel_for(0, devices_count, 1, [&](int & device_id)    {

        
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device_id);
        printf("%s -> unifiedAddressing = %d\n", prop.name, prop.unifiedAddressing);

        hipSetDevice(device_id);

        // hipEvent_t start, stop;
        hipEventCreate(&starts[device_id]);
        hipEventCreate(&stops[device_id]); 

        int is_able;

        hipDeviceCanAccessPeer(&is_able, device_id, device_init_id);
        hipDeviceSynchronize();
        if (is_able)
        { 
            hipDeviceEnablePeerAccess(device_init_id, 0); 
            hipDeviceSynchronize(); 
        }
        else if (device_init_id != device_id)
            printf("Device %i cant access Device %i\n", device_id, device_init_id);

        int range_start  = range*device_id;
        int range_end = range*(device_id + 1);
        printf("device_id: %i [%i, %i)\n", device_id, range_start, range_end);
        

        Aabb * d_boxes_peer;
        hipMalloc((void**)&d_boxes_peer, sizeof(Aabb)*N);
        hipMemcpy(d_boxes_peer, d_boxes, sizeof(Aabb)*N, hipMemcpyDefault);
        hipDeviceSynchronize();  
       
        float3 * d_sm_peer;
        hipMalloc((void**)&d_sm_peer, sizeof(float3)*N);
        hipMemcpy(d_sm_peer, d_sm, sizeof(float3)*N, hipMemcpyDefault);
        hipDeviceSynchronize();  

        MiniBox * d_mini_peer;
        hipMalloc((void**)&d_mini_peer, sizeof(MiniBox)*N);
        hipMemcpy(d_mini_peer, d_mini, sizeof(MiniBox)*N, hipMemcpyDefault);
        hipDeviceSynchronize();  


        hipDeviceCanAccessPeer(&is_able, device_id, device_init_id);
        hipDeviceSynchronize();
        if (is_able)
        { 
            hipDeviceDisablePeerAccess(device_init_id); 
            hipDeviceSynchronize();  
        }
        else if (device_init_id != device_id)
            printf("Device %i cant access Device %i\n", device_id, device_init_id);

        
        // Allocate counter to GPU + set to 0 collisions
        int * d_count;
        gpuErrchk(hipMalloc((void**)&d_count, sizeof(int)));
        gpuErrchk(hipMemset(d_count, 0, sizeof(int)));
        gpuErrchk( hipGetLastError() );   

        // Find overlapping pairs
        int count = 0;
        printf("Guess %i\n", count);

        int2 * d_overlaps;
        hipMalloc((void**)&d_overlaps, sizeof(int2)*(count));
        gpuErrchk( hipGetLastError() ); 
        
        int grid_dim_1d = (range / threads + 1); 
        dim3 grid( grid_dim_1d );

        hipEventRecord(starts[device_id]);
        twostage_queue<<<2*grid_dim_1d, threads>>>(d_sm_peer, d_mini_peer, d_overlaps, N, d_count, count, range_start, range_end);
        hipEventRecord(stops[device_id]);
        hipEventSynchronize(stops[device_id]);
        hipEventElapsedTime(&millisecondss[device_id], starts[device_id], stops[device_id]);
        hipDeviceSynchronize();
        hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
        printf("count for device %i : %i\n", device_id, count);

        if (1){
            printf("Running again\n");
            hipFree(d_overlaps);
            hipMalloc((void**)&d_overlaps, sizeof(int2)*(count));
            // hipMemset(d_overlaps, 0, sizeof(int2)*(count));
            hipMemset(d_count, 0, sizeof(int));
            hipEventRecord(starts[device_id]);
            twostage_queue<<<2*grid_dim_1d, threads>>>(d_sm_peer, d_mini_peer, d_overlaps, N, d_count, count, range_start, range_end);
            hipEventRecord(stops[device_id]);
            hipEventSynchronize(stops[device_id]);
            hipEventElapsedTime(&millisecondss[device_id], starts[device_id], stops[device_id]);
            hipDeviceSynchronize();
            hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
            printf("count2 for device %i : %i\n", device_id, count);
        }
        
        // printf("Elapsed time: %.9f ms/collision\n", milliseconds/count);
        // printf("Boxes: %i\n", N);
        // printf("Elapsed time: %.9f ms/box\n", milliseconds/N);

        // int2 * overlaps = new int2[count];
        int2* overlaps =  (int2*)malloc(sizeof(int2) * count);
        gpuErrchk(hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost));
        gpuErrchk( hipGetLastError() ); 

        printf("Final count for device %i:  %i\n", device_id, count);

        auto& local_overlaps = storages.local();
        // local_overlaps.reserve(local_overlaps.size() + count);
        
        // auto is_face = [&](Aabb x){return x.vertexIds.z >= 0;};
        // auto is_edge = [&](Aabb x){return x.vertexIds.z < 0 && x.vertexIds.y >= 0 ;};
        // auto is_vertex = [&](Aabb x){return x.vertexIds.z < 0  && x.vertexIds.y < 0;};
        
        
        for (size_t i=0; i < count; i++)
        {
            // local_overlaps.emplace_back(overlaps[i].x, overlaps[i].y);
            // finOverlaps.push_back();
            
            int aid = overlaps[i].x;
            int bid = overlaps[i].y;
            Aabb a = boxes[aid];
            Aabb b = boxes[bid];
    
            if (is_vertex(a) && is_face(b)) //vertex, face
                local_overlaps.emplace_back(aid, bid);
            else if (is_edge(a) && is_edge(b))
                local_overlaps.emplace_back(min(aid, bid), max(aid,bid));
            else if (is_face(a) && is_vertex(b))
                local_overlaps.emplace_back(bid, aid);
        }
        
        printf("Total(filt.) overlaps for devid %i: %i\n", device_id, local_overlaps.size());
        // delete [] overlaps;
        // free(overlaps);
        
        // // free(counter);
        // // free(counter);
        // hipFree(d_overlaps);
        // hipFree(d_count); 
        // hipFree(d_b);
        // hipFree(d_r);
        // hipDeviceReset();

    }); //end tbb for loop

    merge_local_overlaps(storages, finOverlaps);

    float longest = 0;
    for (int i=0; i<devices_count; i++)
    {
        for (int j=0; j<devices_count; j++)
        {
            hipEventElapsedTime(&milliseconds, starts[i], stops[j]);
            longest = milliseconds > longest ? milliseconds : longest;
        }

    }
    printf("\n");
    printf("Elapsed time: %.6f ms\n", longest);
    printf("Merged overlaps: %i\n", finOverlaps.size());
    printf("\n");

}

void run_sweep_bigworkerqueue(const Aabb* boxes, int N, int nbox, vector<pair<int, int>>& finOverlaps, int2*& d_overlaps, int *& d_count, int& threads, int & devcount)
{
 
    int device_init_id = 0;

    int smemSize;
    setup(device_init_id, smemSize, threads, nbox);

    hipSetDevice(device_init_id);

    // Allocate boxes to GPU 
    Aabb * d_boxes;
    hipMalloc((void**)&d_boxes, sizeof(Aabb)*N);
    hipMemcpy(d_boxes, boxes, sizeof(Aabb)*N, hipMemcpyHostToDevice);

    // dim3 block(threads);
    int grid_dim_1d = (N / threads + 1); 
    // dim3 grid( grid_dim_1d );
    printf("Grid dim (1D): %i\n", grid_dim_1d);
    printf("Box size: %i\n", sizeof(Aabb));
    // printf("MiniBox size: %i\n", sizeof(MiniBox));
    printf("float3 size: %i\n", sizeof(float3));
    printf("sizeof(queue) size: %i\n", sizeof(Queue));
    
    // float3 * d_sm;
    // hipMalloc((void**)&d_sm, sizeof(float3)*N);

    // MiniBox * d_mini;
    // hipMalloc((void**)&d_mini, sizeof(MiniBox)*N);

    // // mean of all box points (used to find best axis)
    // float3 * d_mean;
    // hipMalloc((void**)&d_mean, sizeof(float3));
    // hipMemset(d_mean, 0, sizeof(float3));

    // // recordLaunch("create_ds", grid_dim_1d, threads, smemSize, create_ds, d_boxes, d_sm, d_mini, N, d_mean);
    // recordLaunch("calc_mean", grid_dim_1d, threads, smemSize, calc_mean, d_boxes, d_mean, N);

    // // temporary
    // float3 mean;
    // hipMemcpy(&mean, d_mean, sizeof(float3), hipMemcpyDeviceToHost);
    // printf("mean: x %.6f y %.6f z %.6f\n", mean.x, mean.y, mean.z);

    // // calculate variance and determine which axis to sort on
    // float3 * d_var; //2 vertices per box
    // hipMalloc((void**)&d_var, sizeof(float3));
    // hipMemset(d_var, 0, sizeof(float3));
    // // calc_variance(boxes, d_var, N, d_mean);
    // recordLaunch("calc_variance", grid_dim_1d, threads, smemSize, calc_variance, d_boxes, d_var, N, d_mean);
    // hipDeviceSynchronize();

    // float3 var3d;
    // hipMemcpy(&var3d, d_var, sizeof(float3), hipMemcpyDeviceToHost);
    // float maxVar = max(max(var3d.x, var3d.y), var3d.z);

    // printf("var: x %.6f y %.6f z %.6f\n", var3d.x, var3d.y, var3d.z);

    Dimension axis;
    // if (maxVar == var3d.x)
    //     axis = x;
    // else if (maxVar == var3d.y)
    //     axis = y;
    // else 
    //     axis = z;
    // //hack
    axis = x;

    printf("Axis: %s\n", axis == x ? "x" : (axis == y ? "y" : "z"));

    // recordLaunch("create_ds", grid_dim_1d, threads, smemSize, create_ds, d_boxes, d_sm, d_mini, N, axis);


    try{
        // thrust::sort(thrust::device, d_sortedmin, d_sortedmin + N, sort_aabb_x() );
        thrust::sort(thrust::device, d_boxes, d_boxes + N, sort_aabb_x());
        }
    catch (thrust::system_error &e){
        printf("Thrust error: %s \n",e.what());}
    
    gpuErrchk( hipGetLastError() );


    int count = 300*N;

    // int * d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    hipMemset(d_count, 0, sizeof(int));

    // int2 * d_overlaps;
    hipMalloc((void**)&d_overlaps, sizeof(int2)*count);

    // create worker queue
    int2 * d_queue;
    // unsigned SIZE = 2000000;
    unsigned SIZE = N;
    hipMalloc((void**)&d_queue, sizeof(int2)*SIZE);

    // start w/ tid, tid + 1
    recordLaunch("init_bigworkerqueue", grid_dim_1d, threads, init_bigworkerqueue, d_queue, N);
    gpuErrchk(hipDeviceSynchronize());

    int start = 0;

    unsigned * d_end;
    hipMalloc((void**)&d_end, sizeof(unsigned));
    hipMemcpy(d_end, &N, sizeof(unsigned), hipMemcpyHostToDevice);

    unsigned end;
    hipMemcpy(&end, d_end, sizeof(int), hipMemcpyDeviceToHost);
    printf("start %i, end %i, N %i\n", start, end, N);

    int TotBoxes = N;

    hipEvent_t b, e;
    hipEventCreate(&b);
    hipEventCreate(&e);

    // Get number of collisions
    hipEventRecord(b);

    int * d_N;
    hipMalloc((void**)&d_N, sizeof(int));
    hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice);
    
    int inc = 0;
    while (N > 0)
    {
        sweepqueue<<<N/threads + 1, threads>>>(d_queue, d_boxes, d_count, count, d_N, N, TotBoxes,  start, d_end, d_overlaps);
        gpuErrchk(hipDeviceSynchronize());
        hipMemcpy(&N, d_N, sizeof(int), hipMemcpyDeviceToHost);
        // hipMemcpy(&end, d_end, sizeof(int), hipMemcpyDeviceToHost);
        // start += N;
        // start = start % 2000000;
        // N = (end - start );
        // N = N < 0 ? end + 2000000 - start  : N;
        
        if (inc % 100)
        // printf("start %i, end %i, N %i\n", start, end, N);
        inc++;
    }
    hipEventRecord(e);
    hipEventSynchronize(e);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, b, e);
    printf("Elapsed time: %.6f ms\n", milliseconds);

    // gpuErrchk(hipDeviceSynchronize());
    // hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    // printf("1st count for device %i:  %i\n", device_init_id, count);

    // hipMalloc((void**)&d_overlaps, sizeof(int2)*(count)); 
    // hipMemset(d_count, 0, sizeof(int));
    // recordLaunch<float3 *, const MiniBox *, int2 *, int, int *, int, int, int>("twostage_queue_2nd", 2*grid_dim_1d, threads,twostage_queue, d_sm, d_mini, d_overlaps, N, d_count, count, 0, INT_MAX);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
    printf("Final count for device %i:  %i\n", device_init_id, count);
    // hipMalloc((void**)&d_overlaps, sizeof(int2)*(count)); //big enough
    // hipMemset(d_count, 0, sizeof(int));
    // retrieve_collision_pairs2<<<grid2, block, 49152>>>(d_boxes, d_count, outpair, d_overlaps, N, count);
    
    int2* overlaps =  (int2*)malloc(sizeof(int2) * count);
    gpuErrchk(hipMemcpy( overlaps, d_overlaps, sizeof(int2)*(count), hipMemcpyDeviceToHost));
    gpuErrchk( hipGetLastError() ); 

    printf("Final count for device %i:  %i\n", 0, count);

    auto& local_overlaps = finOverlaps;
    // local_overlaps.reserve(local_overlaps.size() + count);
    
    hipFree(d_boxes);
    // hipFree(d_mini);
    // hipFree(d_sm);
    
    for (size_t i=0; i < count; i++)
    {
        // local_overlaps.emplace_back(overlaps[i].x, overlaps[i].y);
        // finOverlaps.push_back();
        int aid = overlaps[i].x;
        int bid = overlaps[i].y;
        Aabb a = boxes[aid];
        Aabb b = boxes[bid];

        if (is_vertex(a) && is_face(b)) //vertex, face
            local_overlaps.emplace_back(aid, bid);
        else if (is_edge(a) && is_edge(b))
            local_overlaps.emplace_back(min(aid, bid), max(aid,bid));
        else if (is_face(a) && is_vertex(b))
            local_overlaps.emplace_back(bid, aid);
        // if (is_vertex(a) && is_face(b)) //vertex, face
        // {
        //     local_overlaps.emplace_back(a.ref_id, b.ref_id);
        // }
        // else if (is_face(a) && is_vertex(b))
        // {
        //     local_overlaps.emplace_back(b.ref_id, a.ref_id);
        // }
        // else if (is_edge(a) && is_edge(b))
        // {
        //     local_overlaps.emplace_back(min(a.ref_id, b.ref_id), max(a.ref_id, b.ref_id));
        // }
    }
    free(overlaps);
    printf("Total(filt.) overlaps for devid %i: %i\n", 0, local_overlaps.size());
}