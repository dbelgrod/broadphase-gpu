#include "hip/hip_runtime.h"
#include <gpubf/collision.cuh>

// #include <hip/hip_cooperative_groups.h>
// using namespace cooperative_groups;

using namespace ccdgpu;

__global__ void count_collisions(Aabb * boxes, int * count, int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;
    
    const Aabb& a = boxes[tid];
    const Aabb& b = boxes[tid];

    bool collides = 
        a.max.x >= b.min.x && a.min.x <= b.max.x &&
            a.max.y >= b.min.y && a.min.y <= b.max.y &&
            a.max.z >= b.min.z && a.min.z <= b.max.z;

    if (collides)
        atomicAdd(count, 1);
}

__device__ bool does_collide(const Aabb& a, const Aabb& b)
{
    return 
    //    a.max.x >= b.min.x && a.min.x <= b.max.x &&
            a.max.y >= b.min.y && a.min.y <= b.max.y &&
            a.max.z >= b.min.z && a.min.z <= b.max.z;
}

__device__ bool does_collide(Aabb* a, Aabb* b)
{
    return 
    //    a->max.x >= b->min.x && a->min.x <= b->max.x &&
            a->max.y >= b->min.y && a->min.y <= b->max.y &&
            a->max.z >= b->min.z && a->min.z <= b->max.z;
}

__device__ bool does_collide(const MiniBox& a, const MiniBox& b)
{
    return 
        a.max.x >= b.min.x && a.min.x <= b.max.x &&
     a.max.y >= b.min.y && a.min.y <= b.max.y;
    //  a.max.z >= b.min.z && a.min.z <= b.max.z;

    
}

__device__ bool covertex(const int3& a, const int3& b) {
    
    return a.x == b.x || a.x == b.y || a.x == b.z || 
        a.y == b.x || a.y == b.y || a.y == b.z || 
        a.z == b.x || a.z == b.y || a.z == b.z;
}

// __device__ bool covertex(const float3& a, const float3& b) {
//     return a.x == b.x || a.x == b.y || a.x == b.z || 
//         a.y == b.x || a.y == b.y || a.y == b.z || 
//         a.z == b.x || a.z == b.y || a.z == b.z;
// }

// __device__ bool covertex_box(const Aabb a, const Aabb b) {
//     return a.max.x == b.max.x || a.max.x == b.max.y || a.max.x == b.max.z || 
//         a.max.y == b.max.x || a.max.y == b.max.y || a.max.y == b.max.z || 
//         a.max.z == b.max.x || a.max.z == b.max.y || a.max.z == b.max.z;
// }

__device__ void add_overlap(const int& xid, const int& yid, int * count, int2 * overlaps, int G)
{
    int i = atomicAdd(count, 1);

    if (i < G)
    {
        overlaps[i] = make_int2(xid, yid);
    } 
}

__device__ void append_queue(const int2& lastcheck, int inc, int2 * queue, int * d_N, unsigned * end)
{
    int i = atomicInc(end, *d_N);
    queue[i] = make_int2(lastcheck.x, lastcheck.y + inc);
    
    // queue[i].y += inc;
}

__global__ void get_collision_pairs(Aabb * boxes, int * count, int2 * overlaps, int N, int G, const int nBoxesPerThread, long long * queries)
{       
        extern __shared__ Aabb s_objects[];
        
        Aabb* s_x = &s_objects[0];
        Aabb* s_y = &s_objects[nBoxesPerThread*(BLOCK_PADDED)];


        int threadRowId = nBoxesPerThread*blockIdx.x * blockDim.x + threadIdx.x;
        int threadColId = nBoxesPerThread*blockIdx.y * blockDim.y + threadIdx.y;

        // ex (threadRowId,threadColId) = (0,0) should not be considered but now it contains (1,0) so it must be incl.
        
        //  atomicAdd(queries, 1);
        if (threadRowId >= N || threadColId >= N) return ;//||  threadColId - nBoxesPerThread*blockDim.y >= threadRowId) return;


        // #pragma unroll
        for (int shift = 0; shift < nBoxesPerThread; shift++)
        {
            int tidRow = threadRowId + shift*blockDim.x;
            int xIdx = (shift)*(BLOCK_PADDED) + threadIdx.x;
            // int xIdx = nBoxesPerThread*(threadIdx.x+1) + shift;
            s_x[xIdx]= boxes[tidRow];
            
        
            int tidCol = threadColId + shift*blockDim.y;
            int yIdx = (shift)*(BLOCK_PADDED) + threadIdx.y;
            // int yIdx = nBoxesPerThread*(threadIdx.y+1) + shift;
            s_y[yIdx] = boxes[tidCol];
        }

        // Aabb xboxes [30];
        // Aabb yboxes [30];
        // #pragma unroll 
        for (int i=0; i < nBoxesPerThread; i+=1)
        {
            // #pragma unroll 
            for (int j=0; j < nBoxesPerThread; j+=1)
            {
                //reverse map to global mem
                int g_x__id = threadRowId + i*blockDim.x; 
                // int g_y__id = threadColId + (j-nBoxesPerThread)*blockDim.y; 
                int g_y__id = threadColId + j*blockDim.y; 

                if (g_x__id >= N || g_y__id >= N || g_y__id >= g_x__id) continue;
               

                Aabb * x = &s_x[i*(BLOCK_PADDED) + threadIdx.x];      
                Aabb * y = &s_y[j*(BLOCK_PADDED) + threadIdx.y];

                // Aabb x = boxes[g_x__id];
                // Aabb y = boxes[g_y__id];
            
                atomicAdd((uint*)queries, 1);

                if (
                    does_collide(x,y) &&
                    !covertex(x->vertexIds, y->vertexIds)
                    // !covertex(xmax, ymax) &&
                    // !covertex(xmax, ymin) && 
                    // !covertex(xmin, ymin) &&
                    // !covertex(xmin, ymax)
                    )
                    add_overlap(g_x__id, g_y__id, count, overlaps, G);
            }
        }
    
}

// template<typename T>
__global__ void reset_counter(uint * counter){
    *counter = 0;
}

__global__ void reset_counter(long long * counter)
{
    *counter = 0;
}

__global__ void reset_counter(int * counter){
    printf("Old counter value: %i\n", *counter);
    *counter = 0;
}

__global__ void get_collision_pairs_old(Aabb * boxes, int * count, int2 * overlaps, int N, int G)
{
    
        int threadRowId = blockIdx.x * blockDim.x + threadIdx.x;
        int threadColId = blockIdx.y * blockDim.y + threadIdx.y;
       
        if (threadRowId >= N || threadColId >= N || threadColId >= threadRowId) return;
    
        const Aabb& a = boxes[threadRowId];
        const Aabb& b = boxes[threadColId];
        if ( does_collide(a,b) )
            add_overlap(threadRowId, threadColId, count, overlaps, G);
}
